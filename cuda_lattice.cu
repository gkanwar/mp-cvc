#include "hip/hip_runtime.h"
/* -*- mode: c++ -*- */

#include "cuda_lattice.h"

#include <cassert>
#include <hip/hip_runtime.h>

/// from global.h without pulling in the whole header
#define _GSI(_ix) (24*(_ix))

__device__ __constant__ int gamma_permutation[16][24] = {
  {12, 13, 14, 15, 16, 17, 18, 19, 20, 21, 22, 23, 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11},
  {19, 18, 21, 20, 23, 22, 13, 12, 15, 14, 17, 16, 7, 6, 9, 8, 11, 10, 1, 0, 3, 2, 5, 4},
  {18, 19, 20, 21, 22, 23, 12, 13, 14, 15, 16, 17, 6, 7, 8, 9, 10, 11, 0, 1, 2, 3, 4, 5},
  {13, 12, 15, 14, 17, 16, 19, 18, 21, 20, 23, 22, 1, 0, 3, 2, 5, 4, 7, 6, 9, 8, 11, 10},
  {0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16, 17, 18, 19, 20, 21, 22, 23},
  {0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16, 17, 18, 19, 20, 21, 22, 23},
  {12, 13, 14, 15, 16, 17, 18, 19, 20, 21, 22, 23, 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11},
  {19, 18, 21, 20, 23, 22, 13, 12, 15, 14, 17, 16, 7, 6, 9, 8, 11, 10, 1, 0, 3, 2, 5, 4},
  {18, 19, 20, 21, 22, 23, 12, 13, 14, 15, 16, 17, 6, 7, 8, 9, 10, 11, 0, 1, 2, 3, 4, 5},
  {13, 12, 15, 14, 17, 16, 19, 18, 21, 20, 23, 22, 1, 0, 3, 2, 5, 4, 7, 6, 9, 8, 11, 10},
  {7, 6, 9, 8, 11, 10, 1, 0, 3, 2, 5, 4, 19, 18, 21, 20, 23, 22, 13, 12, 15, 14, 17, 16},
  {6, 7, 8, 9, 10, 11, 0, 1, 2, 3, 4, 5, 18, 19, 20, 21, 22, 23, 12, 13, 14, 15, 16, 17},
  {1, 0, 3, 2, 5, 4, 7, 6, 9, 8, 11, 10, 13, 12, 15, 14, 17, 16, 19, 18, 21, 20, 23, 22},
  {1, 0, 3, 2, 5, 4, 7, 6, 9, 8, 11, 10, 13, 12, 15, 14, 17, 16, 19, 18, 21, 20, 23, 22},
  {6, 7, 8, 9, 10, 11, 0, 1, 2, 3, 4, 5, 18, 19, 20, 21, 22, 23, 12, 13, 14, 15, 16, 17},
  {7, 6, 9, 8, 11, 10, 1, 0, 3, 2, 5, 4, 19, 18, 21, 20, 23, 22, 13, 12, 15, 14, 17, 16}
};
__device__ __constant__ int gamma_sign[16][24] = {
  {-1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
  {+1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1},
  {-1, -1, -1, -1, -1, -1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, -1, -1, -1, -1, -1, -1},
  {+1, -1, +1, -1, +1, -1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, +1, -1, +1, -1, +1, -1},
  {+1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1},
  {+1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
  {+1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
  {-1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1},
  {+1, +1, +1, +1, +1, +1, -1, -1, -1, -1, -1, -1, +1, +1, +1, +1, +1, +1, -1, -1, -1, -1, -1, -1},
  {-1, +1, -1, +1, -1, +1, +1, -1, +1, -1, +1, -1, -1, +1, -1, +1, -1, +1, +1, -1, +1, -1, +1, -1},
  {+1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1},
  {-1, -1, -1, -1, -1, -1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, -1, -1, -1, -1, -1, -1},
  {+1, -1, +1, -1, +1, -1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, +1, -1, +1, -1, +1, -1},
  {-1, +1, -1, +1, -1, +1, +1, -1, +1, -1, +1, -1, -1, +1, -1, +1, -1, +1, +1, -1, +1, -1, +1, -1},
  {-1, -1, -1, -1, -1, -1, +1, +1, +1, +1, +1, +1, -1, -1, -1, -1, -1, -1, +1, +1, +1, +1, +1, +1},
  {-1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1}
};


/**
 * See geometry comment in cuda_lattice.h
 */
const int BS = CUDA_BLOCK_SIZE;
__device__ inline Coord get_thread_origin(Geom local_geom) {
  int x = BS*(blockIdx.x * blockDim.x + threadIdx.x);
  int t = BS*(x / local_geom.LX);
  x %= local_geom.LX;
  int y = BS*(blockIdx.y * blockDim.y + threadIdx.y);
  int z = BS*(blockIdx.z * blockDim.z + threadIdx.z);
  return Coord { .t = t, .x = x, .y = y, .z = z };
}
__device__ inline size_t coord2lexic(Coord coord, Geom local_geom) {
  return (((coord.t*local_geom.LX) + coord.x)*local_geom.LY + coord.y)*local_geom.LZ + coord.z;
}

/**
 * Given length-24 spin vector in, multiply by appropriate gamma matrix, writing
 * to out (non-aliasing assumed).
 */
__device__ inline void _fv_eq_gamma_ti_fv(double* out, int gamma_index, const double* in) {
  for (int i = 0; i < 24; ++i) {
    out[i] = in[gamma_permutation[gamma_index][i]] * gamma_sign[gamma_index][i];
  }
}
__device__ inline void _fv_ti_eq_g5(double* in_out) {
  for (int i = 12; i < 24; ++i) {
    in_out[i] *= -1;
  }
}

/**
 * 1D kernels: operate over CUDA_BLOCK_SIZE spinor elements each.
 *  - `len`: num *doubles* in the input/output array (must be divisible by 24)
 */
__global__ void ker_spinor_field_eq_gamma_ti_spinor_field(
    double* out, int gamma_index, const double* in, size_t len) {
  int start_ind = BS*_GSI(blockIdx.x * blockDim.x + threadIdx.x);
  for (int ind = start_ind; ind < len && ind < (start_ind + BS*24); ind += 24) {
    double* rr = out + ind;
    const double* ss = in + ind;
    _fv_eq_gamma_ti_fv(rr, gamma_index, ss);
  }
}

__global__ void ker_g5_phi(double* spinor, size_t len) {
  /* invert sign of spin components 2 and 3 */
  int start_ind = BS*_GSI(blockIdx.x * blockDim.x + threadIdx.x);
  for (int ind = start_ind; ind < len && ind < (start_ind + BS*24); ind += 24) {
    _fv_ti_eq_g5(&spinor[ind]);
  }
}

// __device__ void site_map(int xv[4], int const x[4], Geom global_geom ) {
//   int T_global = global_geom.T;
//   int LX_global = global_geom.LX;
//   int LY_global = global_geom.LY;
//   int LZ_global = global_geom.LZ;
//   xv[0] = ( x[0] >= T_global   / 2 ) ? (x[0] - T_global )  : x[0];
//   xv[1] = ( x[1] >= LX_global  / 2 ) ? (x[1] - LX_global)  : x[1];
//   xv[2] = ( x[2] >= LY_global  / 2 ) ? (x[2] - LY_global)  : x[2];
//   xv[3] = ( x[3] >= LZ_global  / 2 ) ? (x[3] - LZ_global)  : x[3];
// }

// __device__ void site_map_zerohalf (int xv[4], int const x[4], Geom global_geom ) {
//   int T_global = global_geom.T;
//   int LX_global = global_geom.LX;
//   int LY_global = global_geom.LY;
//   int LZ_global = global_geom.LZ;
//   xv[0] = ( x[0] > T_global   / 2 ) ? x[0] - T_global   : (  ( x[0] < T_global   / 2 ) ? x[0] : 0 );
//   xv[1] = ( x[1] > LX_global  / 2 ) ? x[1] - LX_global  : (  ( x[1] < LX_global  / 2 ) ? x[1] : 0 );
//   xv[2] = ( x[2] > LY_global  / 2 ) ? x[2] - LY_global  : (  ( x[2] < LY_global  / 2 ) ? x[2] : 0 );
//   xv[3] = ( x[3] > LZ_global  / 2 ) ? x[3] - LZ_global  : (  ( x[3] < LZ_global  / 2 ) ? x[3] : 0 );
// }

__device__ int coord_map_zerohalf(int xi, int Li) {
  return (xi > Li / 2) ? xi - Li : ( (xi < Li / 2) ? xi : 0 );
}


/**
 * 4D kernels: operate over CUDA_BLOCK_SIZE^4 spinor elements each.
 */
__global__ void ker_dzu_dzsu(
    double* d_dzu, double* d_dzsu, const double* fwd_src, const double* fwd_y,
    int iflavor, Coord g_proc_coords, Coord gsx, IdxComb idx_comb,
    Geom global_geom, Geom local_geom) {
  Coord origin = get_thread_origin(local_geom);
  int gsx_arr[4] = {gsx.t, gsx.x, gsx.y, gsx.z};
  size_t VOLUME = local_geom.T * local_geom.LX * local_geom.LY * local_geom.LZ;
  int local_geom_arr[4] = {local_geom.T, local_geom.LX, local_geom.LY, local_geom.LZ};
  int global_geom_arr[4] = {global_geom.T, global_geom.LX, global_geom.LY, global_geom.LZ};
  int proc_coord_arr[4] = {g_proc_coords.t, g_proc_coords.x, g_proc_coords.y, g_proc_coords.z};
  double dzu_work[6 * 12 * 12 * 2] = { 0 };
  double dzsu_work[6 * 12 * 12 * 2] = { 0 };
  double spinor_work_0[24] = { 0 };
  double spinor_work_1[24] = { 0 };
  for (int ia = 0; ia < 12; ++ia) {
    for (int k = 0; k < 6; ++k) {
      const int sigma = idx_comb.comb[k][1];
      const int rho = idx_comb.comb[k][0];
      const double* fwd_base = &fwd_src[_GSI(VOLUME) * (iflavor * 12 + ia)];
      for (int dt = 0; dt < BS; ++dt) {
        for (int dx = 0; dx < BS; ++dx) {
          for (int dy = 0; dy < BS; ++dy) {
            for (int dz = 0; dz < BS; ++dz) {
              const int tt = dt + origin.t;
              const int xx = dx + origin.x;
              const int yy = dy + origin.y;
              const int zz = dz + origin.z;
              if (tt >= local_geom.T || xx >= local_geom.LX ||
                  yy >= local_geom.LY || zz >= local_geom.LZ) {
                continue;
              }
              const Coord coord{
                .t = tt, .x = xx, .y = yy, .z = zz
              };
              size_t iz = coord2lexic(coord, local_geom);
              const double* _u = &fwd_base[_GSI(iz)];
              double* _t_sigma = spinor_work_0;
              double* _t_rho = spinor_work_1;
              _fv_eq_gamma_ti_fv(_t_sigma, sigma, _u);
              _fv_ti_eq_g5(_t_sigma);
              _fv_eq_gamma_ti_fv(_t_rho, rho, _u);
              _fv_ti_eq_g5(_t_rho);
              int coord_arr[4] = {tt, xx, yy, zz};
              int zrho = coord_arr[rho] + proc_coord_arr[rho] * local_geom_arr[rho] - gsx_arr[rho];
              zrho = (zrho + global_geom_arr[rho]) % global_geom_arr[rho];
              int zsigma = coord_arr[sigma] + proc_coord_arr[sigma] * local_geom_arr[sigma] - gsx_arr[sigma];
              zsigma = (zsigma + global_geom_arr[sigma]) % global_geom_arr[sigma];
              int factor_rho = coord_map_zerohalf(zrho, global_geom_arr[rho]);
              int factor_sigma = coord_map_zerohalf(zsigma, global_geom_arr[sigma]);
              for (int ib = 0; ib < 12; ++ib) {
                for (int i = 0; i < 12; ++i) {
                  double fwd_y_re = fwd_y[((1-iflavor) * 12 + ib) * _GSI(VOLUME) + _GSI(iz) + 2*i];
                  double fwd_y_im = fwd_y[((1-iflavor) * 12 + ib) * _GSI(VOLUME) + _GSI(iz) + 2*i+1];
                  double s_re = (_t_sigma[2*i] * factor_rho - _t_rho[2*i] * factor_sigma);
                  double s_im = (_t_sigma[2*i+1] * factor_rho - _t_rho[2*i+1] * factor_sigma);
                  dzu_work[((k * 12 + ia) * 12 + ib) * 2 + 0] += fwd_y_re * s_re + fwd_y_im * s_im;
                  dzu_work[((k * 12 + ia) * 12 + ib) * 2 + 1] += fwd_y_re * s_im - fwd_y_im * s_re;
                }
              }
            }
          }
        }
      } // end vol loop

      // reduce (TODO faster reduce algo?)
      for (int ib = 0; ib < 12; ++ib) {
        int ind = ((k * 12 + ia) * 12 + ib) * 2;
        atomicAdd_system(&d_dzu[ind], dzu_work[ind]);
        atomicAdd_system(&d_dzu[ind+1], dzu_work[ind+1]);
      }
    }

    for (int sigma = 0; sigma < 4; ++sigma) {
      const double* fwd_base = &fwd_src[_GSI(VOLUME) * (iflavor * 12 + ia)];
      for (int ib = 0; ib < 12; ++ib) {
        for (int dt = 0; dt < BS; ++dt) {
          for (int dx = 0; dx < BS; ++dx) {
            for (int dy = 0; dy < BS; ++dy) {
              for (int dz = 0; dz < BS; ++dz) {
                const int tt = dt + origin.t;
                const int xx = dx + origin.x;
                const int yy = dy + origin.y;
                const int zz = dz + origin.z;
                if (tt >= local_geom.T || xx >= local_geom.LX ||
                    yy >= local_geom.LY || zz >= local_geom.LZ) {
                  continue;
                }
                const Coord coord{
                  .t = tt, .x = xx, .y = yy, .z = zz
                };
                size_t iz = coord2lexic(coord, local_geom);
                const double* _u = &fwd_base[_GSI(iz)];
                double* _t = spinor_work_0;
                _fv_eq_gamma_ti_fv(_t, sigma, _u);
                _fv_ti_eq_g5(_t);

                for (int i = 0; i < 12; ++i) {
                  double fwd_y_re = fwd_y[((1-iflavor) * 12 + ib) * _GSI(VOLUME) + _GSI(iz) + 2*i];
                  double fwd_y_im = fwd_y[((1-iflavor) * 12 + ib) * _GSI(VOLUME) + _GSI(iz) + 2*i+1];
                  double s_re = _t[2*i];
                  double s_im = _t[2*i+1];
                  dzsu_work[((sigma * 12 + ia) * 12 + ib) * 2 + 0] += fwd_y_re * s_re + fwd_y_im * s_im;
                  dzsu_work[((sigma * 12 + ia) * 12 + ib) * 2 + 1] += fwd_y_re * s_im - fwd_y_im * s_re;
                }
              }
            }
          }
        } // end vol loop

        // reduce (TODO faster reduce algo?)
        int ind = ((sigma * 12 + ia) * 12 + ib) * 2;
        atomicAdd_system(&d_dzsu[ind], dzsu_work[ind]);
        atomicAdd_system(&d_dzsu[ind+1], dzsu_work[ind+1]);
      }
    }
  }
}

/**
 * Top-level operations.
 */
void cu_spinor_field_eq_gamma_ti_spinor_field(double* out, int mu, const double* in, size_t len) {
  const size_t BS_spinor = 12 * CUDA_BLOCK_SIZE * CUDA_THREAD_DIM_1D;
  size_t nx = (len + BS_spinor - 1) / BS_spinor;
  dim3 kernel_nblocks(nx);
  dim3 kernel_nthreads(CUDA_THREAD_DIM_1D);
  ker_spinor_field_eq_gamma_ti_spinor_field<<<kernel_nblocks, kernel_nthreads>>>(
      out, mu, in, len);
}

void cu_g5_phi(double* spinor, size_t len) {
  const size_t BS_spinor = 12 * CUDA_BLOCK_SIZE * CUDA_THREAD_DIM_1D;
  size_t nx = (len + BS_spinor - 1) / BS_spinor;
  dim3 kernel_nblocks(nx);
  dim3 kernel_nthreads(CUDA_THREAD_DIM_1D);
  ker_g5_phi<<<kernel_nblocks, kernel_nthreads>>>(spinor, len);
}

void cu_dzu_dzsu(
    double* d_dzu, double* d_dzsu, const double* fwd_src, const double* fwd_y,
    int iflavor, Coord proc_coords, Coord gsx, IdxComb idx_comb, Geom global_geom, Geom local_geom) {
  int T = local_geom.T;
  int LX = local_geom.LX;
  int LY = local_geom.LY;
  int LZ = local_geom.LZ;
  const size_t BS_TX = CUDA_THREAD_DIM_4D * CUDA_BLOCK_SIZE * CUDA_BLOCK_SIZE;
  const size_t BS_Y = CUDA_THREAD_DIM_4D * CUDA_BLOCK_SIZE;
  const size_t BS_Z = CUDA_THREAD_DIM_4D * CUDA_BLOCK_SIZE;
  size_t nx = (T*LX + BS_TX - 1) / BS_TX;
  size_t ny = (LY + BS_Y - 1) / BS_Y;
  size_t nz = (LZ + BS_Z - 1) / BS_Z;
  dim3 kernel_nblocks(nx, ny, nz);
  dim3 kernel_nthreads(CUDA_THREAD_DIM_4D, CUDA_THREAD_DIM_4D, CUDA_THREAD_DIM_4D);
  ker_dzu_dzsu<<<kernel_nblocks, kernel_nthreads>>>(
      d_dzu, d_dzsu, fwd_src, fwd_y, iflavor, proc_coords, gsx, idx_comb,
      global_geom, local_geom);
}


