#include "hip/hip_runtime.h"
/* -*- mode: c++ -*- */

#include "cuda_lattice.h"

__device__ __constant__ int gamma_permutation[16][24] = {
  {12, 13, 14, 15, 16, 17, 18, 19, 20, 21, 22, 23, 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11},
  {19, 18, 21, 20, 23, 22, 13, 12, 15, 14, 17, 16, 7, 6, 9, 8, 11, 10, 1, 0, 3, 2, 5, 4},
  {18, 19, 20, 21, 22, 23, 12, 13, 14, 15, 16, 17, 6, 7, 8, 9, 10, 11, 0, 1, 2, 3, 4, 5},
  {13, 12, 15, 14, 17, 16, 19, 18, 21, 20, 23, 22, 1, 0, 3, 2, 5, 4, 7, 6, 9, 8, 11, 10},
  {0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16, 17, 18, 19, 20, 21, 22, 23},
  {0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16, 17, 18, 19, 20, 21, 22, 23},
  {12, 13, 14, 15, 16, 17, 18, 19, 20, 21, 22, 23, 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11},
  {19, 18, 21, 20, 23, 22, 13, 12, 15, 14, 17, 16, 7, 6, 9, 8, 11, 10, 1, 0, 3, 2, 5, 4},
  {18, 19, 20, 21, 22, 23, 12, 13, 14, 15, 16, 17, 6, 7, 8, 9, 10, 11, 0, 1, 2, 3, 4, 5},
  {13, 12, 15, 14, 17, 16, 19, 18, 21, 20, 23, 22, 1, 0, 3, 2, 5, 4, 7, 6, 9, 8, 11, 10},
  {7, 6, 9, 8, 11, 10, 1, 0, 3, 2, 5, 4, 19, 18, 21, 20, 23, 22, 13, 12, 15, 14, 17, 16},
  {6, 7, 8, 9, 10, 11, 0, 1, 2, 3, 4, 5, 18, 19, 20, 21, 22, 23, 12, 13, 14, 15, 16, 17},
  {1, 0, 3, 2, 5, 4, 7, 6, 9, 8, 11, 10, 13, 12, 15, 14, 17, 16, 19, 18, 21, 20, 23, 22},
  {1, 0, 3, 2, 5, 4, 7, 6, 9, 8, 11, 10, 13, 12, 15, 14, 17, 16, 19, 18, 21, 20, 23, 22},
  {6, 7, 8, 9, 10, 11, 0, 1, 2, 3, 4, 5, 18, 19, 20, 21, 22, 23, 12, 13, 14, 15, 16, 17},
  {7, 6, 9, 8, 11, 10, 1, 0, 3, 2, 5, 4, 19, 18, 21, 20, 23, 22, 13, 12, 15, 14, 17, 16}
};
__device__ __constant__ int gamma_sign[16][24] = {
  {-1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
  {+1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1},
  {-1, -1, -1, -1, -1, -1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, -1, -1, -1, -1, -1, -1},
  {+1, -1, +1, -1, +1, -1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, +1, -1, +1, -1, +1, -1},
  {+1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1},
  {+1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
  {+1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
  {-1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1},
  {+1, +1, +1, +1, +1, +1, -1, -1, -1, -1, -1, -1, +1, +1, +1, +1, +1, +1, -1, -1, -1, -1, -1, -1},
  {-1, +1, -1, +1, -1, +1, +1, -1, +1, -1, +1, -1, -1, +1, -1, +1, -1, +1, +1, -1, +1, -1, +1, -1},
  {+1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1},
  {-1, -1, -1, -1, -1, -1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, +1, -1, -1, -1, -1, -1, -1},
  {+1, -1, +1, -1, +1, -1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, +1, -1, +1, -1, +1, -1},
  {-1, +1, -1, +1, -1, +1, +1, -1, +1, -1, +1, -1, -1, +1, -1, +1, -1, +1, +1, -1, +1, -1, +1, -1},
  {-1, -1, -1, -1, -1, -1, +1, +1, +1, +1, +1, +1, -1, -1, -1, -1, -1, -1, +1, +1, +1, +1, +1, +1},
  {-1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1, -1, +1}
};


/**
 * See geometry comment in cuda_lattice.h
 */
const int BS = CUDA_BLOCK_SIZE;
__device__ inline Coord get_thread_origin(Geom local_geom) {
  int x = BS*(blockIdx.x * blockDim.x + threadIdx.x);
  int y = BS*(blockIdx.y * blockDim.y + threadIdx.y);
  int z = BS*(blockIdx.z * blockDim.z + threadIdx.z);
  int t = BS*(x / local_geom.X);
  x %= local_geom.X;
}


__global__ void cu_spinor_field_eq_gamma_ti_spinor_field(double* out, const double* in, int gamma_index, size_t len) {
  int start_ind = BS*_GSI(blockIdx.x * blockDim.x + threadIdx.x);
  for (int ind = start_ind; ind < len && ind < (start_ind + BS*24); ind += 24) {
    double* rr = out + ind;
    double* ss = in + ind;
    for (int i = 0; i < 24; ++i) {
      rr[i] = ss[gamma_permutation[gamma_index][i]] * gamma_sign[gamma_index][i];
    }
  }
}

__global__ void cu_g5_phi(double* spinor, size_t len) {
  /* invert sign of spin components 2 and 3 */
  int start_ind = BS*_GSI(blockIdx.x * blockDim.x + threadIdx.x);
  for (int ind = start_ind; ind < len && ind < (start_ind + BS*24); ind += 24) {
    for (int i = 12; i < 24; ++i) {
      spinor[ind + i] *= -1;
    }
  }
}

__device__ void site_map(int xv[4], int const x[4], Geom global_geom ) {
  int T_global = global_geom.T;
  int LX_global = global_geom.LX;
  int LY_global = global_geom.LY;
  int LZ_global = global_geom.LZ;
  xv[0] = ( x[0] >= T_global   / 2 ) ? (x[0] - T_global )  : x[0];
  xv[1] = ( x[1] >= LX_global  / 2 ) ? (x[1] - LX_global)  : x[1];
  xv[2] = ( x[2] >= LY_global  / 2 ) ? (x[2] - LY_global)  : x[2];
  xv[3] = ( x[3] >= LZ_global  / 2 ) ? (x[3] - LZ_global)  : x[3];
}

__device__ void site_map_zerohalf (int xv[4], int const x[4], Geom global_geom ) {
  int T_global = global_geom.T;
  int LX_global = global_geom.LX;
  int LY_global = global_geom.LY;
  int LZ_global = global_geom.LZ;
  xv[0] = ( x[0] > T_global   / 2 ) ? x[0] - T_global   : (  ( x[0] < T_global   / 2 ) ? x[0] : 0 );
  xv[1] = ( x[1] > LX_global  / 2 ) ? x[1] - LX_global  : (  ( x[1] < LX_global  / 2 ) ? x[1] : 0 );
  xv[2] = ( x[2] > LY_global  / 2 ) ? x[2] - LY_global  : (  ( x[2] < LY_global  / 2 ) ? x[2] : 0 );
  xv[3] = ( x[3] > LZ_global  / 2 ) ? x[3] - LZ_global  : (  ( x[3] < LZ_global  / 2 ) ? x[3] : 0 );
}


__global__ void cu_dzu_dzsu(
    double* dzu, double* dzsu, int* d_lexic2coords, Coord g_proc_coords,
    IdxComb idx_comb, Geom global_geom, Geom local_geom) {
  
}
